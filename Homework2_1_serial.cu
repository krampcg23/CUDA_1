
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>

using namespace std;

int countOnes(int** A, int row, int col) {
    int count = 0;
    for (int i = 0; i < row; i++) {
        for (int j = 0; j < col; j++) {
            if (A[i][j] == 1) count++;
        }
    }
    return count;
}

int main(int argc, char* argv[]) {
    if (argc != 2) {
        cerr << "Arguments error" << endl;
        return -1;
    }
    ifstream file(argv[1]);
    int row, col;
    file >> row >> col;

    int** A = new int*[row];
    for (int i = 0; i < row; i++) A[i] = new int[col];

    for (int i = 0; i < row; i++) {
        for (int j = 0; j < col; j++) {
            int element;
            file >> element;
            A[i][j] = element;
        }
    }
    
    file.close();

    int count = countOnes(A, row, col);

    cout << "There are " << count << "  ones in this matrix" << endl;

    for(int i = 0; i < row; ++i) {
            delete [] A[i];
    }
    delete [] A;

    return 0;
}
