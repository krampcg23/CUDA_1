
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>

using namespace std;

int countOnes(int** A, int row, int col) {
    int count = 0;
    for (int i = 0; i < row; i++) {
        for (int j = 0; j < col; j++) {
            if (A[i][j] == 1) count++;
        }
    }
    return count;
}

int main(int argc, char* argv[]) {
    if (argc != 2) {
        cerr << "Arguments error" << endl;
        return -1;
    }
    ifstream file(argv[1]);
    int row, col;
    file >> row >> col;

    int** A = new int*[row];
    A[0] = new int[row * col];
    for (int i = 1; i < row; ++i) A[i] = A[i-1] + col;

    for (int i = 0; i < row; i++) {
        for (int j = 0; j < col; j++) {
            int element;
            file >> element;
            A[i][j] = element;
        }
    }

    file.close();

    int count = countOnes(A, row, col);

    cout << "There are " << count << "  ones in this matrix" << endl;

    delete A[0];
    delete A;

    return 0;
}
