
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>

using namespace std;


__global__ void transpose(int* A, int* B, int row, int col) {

    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= row || j >= col) return;
    B[j*row + i] = A[i*col + j];
}

int main(int argc, char* argv[]) {
    if (argc != 2) {
        cerr << "Arguments error" << endl;
        return -1;
    }
    ifstream file(argv[1]);
    int row, col;
    file >> col >> row;

    int** A = new int*[row];
    A[0] = new int[row*col];
    for (int i = 1; i < row; i++) A[i] = A[i-1] + col;

    for (int i = 0; i < row; i++) {
        for (int j = 0; j < col; j++) {
            int element;
            file >> element;
            A[i][j] = element;
        }
    }
    
    file.close();

    int* count = new int;
    *count = 0;

    int* deviceA;
    int bytes = row * col * sizeof(int);
    hipMalloc(&deviceA, bytes);
    hipMemcpy(deviceA, A[0], bytes, hipMemcpyHostToDevice);

    int** B = new int*[col];
    B[0] = new int[row*col];
    for (int i = 1; i < col; i++) B[i] = B[i-1] + row;
    
    int* deviceB;
    hipMalloc(&deviceB, bytes);

    dim3 threadsPerBlock(8, 8, 1);
    dim3 numBlocks((col + threadsPerBlock.x-1) / threadsPerBlock.x,
                   (row + threadsPerBlock.y-1) / threadsPerBlock.y, 1);

    transpose<<<numBlocks, threadsPerBlock>>>(deviceA, deviceB, row, col);
    //cudaDeviceSynchronize();

    hipMemcpy(B[0], deviceB, bytes,  hipMemcpyDeviceToHost);
    cout << row << " " << col << endl;
    for (int i = 0; i < col; i++) {
        for (int j = 0; j < row; j++) {
            cout << B[i][j] << " ";
        }
        cout << endl;
    }

    delete A[0];
    delete A;
    delete B[0];
    delete B;

    hipFree(deviceA);
    hipFree(deviceB);

    return 0;
}
