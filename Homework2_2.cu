/* CSCI 563 Programming Assignment 2 Part 2
   Clayton Kramp
*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>

using namespace std;

// Device function to transpose matrix
__global__ void transpose(int* A, int* B, int row, int col) {

    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= row || j >= col) return;
    B[j*row + i] = A[i*col + j];
}

int main(int argc, char* argv[]) {
    if (argc != 2) {
        cerr << "Arguments error" << endl;
        return -1;
    }
    ifstream file(argv[1]);
    if (!file.good()) {
        cerr << "Bad input" << endl;
        return -1;
    }
    int row, col;
    file >> col >> row;

    int** A = new int*[row];
    A[0] = new int[row*col];
    for (int i = 1; i < row; i++) A[i] = A[i-1] + col;

    // Fill in matrix A in host
    for (int i = 0; i < row; i++) {
        for (int j = 0; j < col; j++) {
            int element;
            file >> element;
            A[i][j] = element;
        }
    }
    
    file.close();

    int* count = new int;
    *count = 0;

    // Copy matrix to device memory
    int* deviceA;
    int bytes = row * col * sizeof(int);
    hipMalloc(&deviceA, bytes);
    hipMemcpy(deviceA, A[0], bytes, hipMemcpyHostToDevice);
    
    // Create the fill in matrix
    int** B = new int*[col];
    B[0] = new int[row*col];
    for (int i = 1; i < col; i++) B[i] = B[i-1] + row;
    
    int* deviceB;
    hipMalloc(&deviceB, bytes);

    dim3 threadsPerBlock(8, 8, 1);
    dim3 numBlocks((col + threadsPerBlock.x-1) / threadsPerBlock.x,
                   (row + threadsPerBlock.y-1) / threadsPerBlock.y, 1);

    // Call the actual function
    transpose<<<numBlocks, threadsPerBlock>>>(deviceA, deviceB, row, col);
    //cudaDeviceSynchronize();

    // Copy back the memory
    hipMemcpy(B[0], deviceB, bytes,  hipMemcpyDeviceToHost);

    // Print out the info to console
    cout << row << " " << col << endl;
    for (int i = 0; i < col; i++) {
        for (int j = 0; j < row; j++) {
            cout << B[i][j] << " ";
        }
        cout << endl;
    }

    delete A[0];
    delete A;
    delete B[0];
    delete B;

    hipFree(deviceA);
    hipFree(deviceB);

    return 0;
}
