/* CSCI 563 Programming Assignment 2
   Clayton Kramp
*/


#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>

using namespace std;

// Main Device Function to be used to count number of ones
__global__ void countOnes(int* A, int* count, int row, int col) {

    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= row || j >= col) return;
    if (A[i * col + j] == 1) {
        // Atomic addition for race conditions
        atomicAdd(count, 1);
    }
}

int main(int argc, char* argv[]) {
    if (argc != 2) {
        cerr << "Arguments error" << endl;
        return -1;
    }
    ifstream file(argv[1]);
    if (!file.good()) {
        cerr << "Bad input" << endl;
        return -1;
    }
    int row, col;
    file >> col >> row;

    int** A = new int*[row];
    A[0] = new int[row*col];
    for (int i = 1; i < row; i++) A[i] = A[i-1] + col;

    // Fill in Host Array A
    for (int i = 0; i < row; i++) {
        for (int j = 0; j < col; j++) {
            int element;
            file >> element;
            A[i][j] = element;
        }
    }
    
    file.close();

    int* count = new int;
    *count = 0;

    // Copy memory to device array deviceA
    int* deviceA;
    int bytes = row * col * sizeof(int);
    hipMalloc(&deviceA, bytes);
    hipMemcpy(deviceA, A[0], bytes, hipMemcpyHostToDevice);

    // Copy deviceCount
    int* deviceCount;
    hipMalloc(&deviceCount, 4);
    hipMemcpy(deviceCount, count, 4, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(8, 8, 1);
    dim3 numBlocks((col + threadsPerBlock.x-1) / threadsPerBlock.x,
                   (row + threadsPerBlock.y-1) / threadsPerBlock.y, 1);

    // Launch the program
    countOnes<<<numBlocks, threadsPerBlock>>>(deviceA, deviceCount, row, col);
    //cudaDeviceSynchronize();

    // Copy back from device the deviceCount
    hipMemcpy(count, deviceCount, 4, hipMemcpyDeviceToHost);

    cout << *count << endl;

    delete A[0];
    delete A;

    hipFree(deviceA);
    hipFree(deviceCount);

    return 0;
}
