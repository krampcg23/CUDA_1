
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>

using namespace std;


__global__ void countOnes(int* A, int* count, int row, int col) {

    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= row || j >= col) return;
    if (A[i * col + j] == 1) {
        atomicAdd(count, 1);
    }
}

int main(int argc, char* argv[]) {
    if (argc != 2) {
        cerr << "Arguments error" << endl;
        return -1;
    }
    ifstream file(argv[1]);
    int row, col;
    file >> col >> row;

    int** A = new int*[row];
    A[0] = new int[row*col];
    for (int i = 1; i < row; i++) A[i] = A[i-1] + col;

    for (int i = 0; i < row; i++) {
        for (int j = 0; j < col; j++) {
            int element;
            file >> element;
            A[i][j] = element;
        }
    }
    
    file.close();

    int* count = new int;
    *count = 0;

    int* deviceA;
    int bytes = row * col * sizeof(int);
    hipMalloc(&deviceA, bytes);
    hipMemcpy(deviceA, A[0], bytes, hipMemcpyHostToDevice);

    int* deviceCount;
    hipMalloc(&deviceCount, 4);
    hipMemcpy(deviceCount, count, 4, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(8, 8, 1);
    dim3 numBlocks((col + threadsPerBlock.x-1) / threadsPerBlock.x,
                   (row + threadsPerBlock.y-1) / threadsPerBlock.y, 1);

    countOnes<<<numBlocks, threadsPerBlock>>>(deviceA, deviceCount, row, col);
    //cudaDeviceSynchronize();

    hipMemcpy(count, deviceCount, 4, hipMemcpyDeviceToHost);

    cout << "There are " << *count << " ones in this matrix" << endl;

    delete A[0];
    delete A;

    hipFree(deviceA);
    hipFree(deviceCount);

    return 0;
}
